/*

Today's goal is to visualize a simple raytracer::triangle with no lighting

*/

//---------------------------------------------------------

#include <cstdlib>
#define CL_RUN(fnc, ...)                                                       \
  fnc(__VA_ARGS__, &err);                                                      \
  if (err < 0) {                                                               \
    printf("CL ERROR AT LINE %d\n", __LINE__);                                 \
    perror("err");                                                             \
    exit(1);                                                                   \
  }

#define CL_RUN3(exp)                                                           \
  exp;                                                                         \
  if (err < 0) {                                                               \
    printf("CL ERROR AT LINE %d, err: %d\n", __LINE__, err);                   \
    perror("err");                                                             \
    exit(1);                                                                   \
  }

#define CL_RUN2(fnc)                                                           \
  do {                                                                         \
    cl_int err = fnc;                                                          \
    if (err < 0) {                                                             \
      printf("CL ERROR AT LINE %d %d\n", __LINE__, err);                       \
      perror("err");                                                           \
      exit(1);                                                                 \
    }                                                                          \
  } while (0);
//---------------------------------------------------------

// C standard includes
#include <stdio.h>

// OpenCL includes
#include <hip/hip_runtime.h>

#include <math.h>
#include <png.h>
#include <stdbool.h>
#include <stdlib.h>
#include <string.h>

#define NUM_PIXELS_X 32
#define NUM_PIXELS_Y 32
#define OUT_BUFFER_LEN (NUM_PIXELS_X * NUM_PIXELS_Y)

typedef unsigned char uchar;

//-----------------------PNG-----------------------------

void save_png(const char *filename, unsigned char *data, int width,
              int height) {
  FILE *fp = fopen(filename, "wb");
  if (!fp) {
    fprintf(stderr, "Error: Unable to open file %s for writing\n", filename);
    return;
  }

  png_structp png_ptr =
      png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if (!png_ptr) {
    fprintf(stderr, "Error: png_create_write_struct failed\n");
    fclose(fp);
    return;
  }

  png_infop info_ptr = png_create_info_struct(png_ptr);
  if (!info_ptr) {
    fprintf(stderr, "Error: png_create_info_struct failed\n");
    png_destroy_write_struct(&png_ptr, NULL);
    fclose(fp);
    return;
  }

  png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGBA,
               PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_DEFAULT,
               PNG_FILTER_TYPE_DEFAULT);

  png_bytep *row_pointers = (png_bytep *)malloc(sizeof(png_bytep) * height);
  for (int y = 0; y < height; y++) {
    row_pointers[y] =
        (png_bytep)&data[y * width * 4]; // Assuming 4 channels (raytracer::RGB)
  }

  png_init_io(png_ptr, fp);
  png_set_rows(png_ptr, info_ptr, row_pointers);
  png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);

  // Cleanup
  png_destroy_write_struct(&png_ptr, &info_ptr);
  fclose(fp);
  free(row_pointers);
}

float my_ceil(double x);
float my_ceil(double x) {
  int i = (int)x;
  return (x > (double)i) ? (double)(i + 1) : (double)i;
}
float my_floor(double x);
float my_floor(double x) {
  int i = (int)x;
  return (x < (double)i) ? (double)(i - 1) : (double)i;
}
double random_double_between_0_1_giver(uint *seed);
double random_double_between_0_1_giver(uint *seed) {
  const uint a = 1664525;
  const uint c = 1013904223;
  const uint m = 0xFFFFFFFF;

  // Initialize the PRNG with the given seed
  uint state = *seed;

  state = (a * state + c) & m;
  *seed = state;

  return (double)(state) / (double)(m);
  ;
}
namespace raytracer {
typedef struct {
  uchar R;
  uchar G;
  uchar B;
} RGB;

typedef struct {
  double x;
  double y;
} Vector2D;

typedef struct {
  double x;
  double y;
  double z;
} Vector3D;

typedef struct {
  Vector2D *array;
  int number_of_elements;
} Vector2D_array;

typedef struct {
  Vector3D array[50 * 50];
  int number_of_elements;
} Vector3D_array;

#define SPECTRUM_SIZE (((700 - 380) / 5 + 1))

typedef struct {
  double spectrum[(int)SPECTRUM_SIZE];
  int number_of_elements;

} spectrum_of_light;

typedef struct {
  raytracer::spectrum_of_light surface_spectrum;
  double diffusion_coefficient;

} surface;

typedef struct {
  raytracer::surface surface_array[50]; // we have at maximum 50 different
                                        // raytracer::surface propoerties

} surface_array;

typedef struct {
  Vector3D coordinates1;
  Vector3D coordinates2;
  Vector3D coordinates3;
  int surface_number;

} triangle;

typedef struct {
  triangle all_triangles[2000]; // we have at maximum 2000 distinct
                                // raytracer::triangles
  int total_number_of_triangles;

} triangle_array;

typedef struct {
  Vector3D ray_current_coordinates;
  Vector3D ray_direction;
  raytracer::spectrum_of_light ray_spectrum;
  int recursion_index;

} ray;

} // namespace raytracer

typedef raytracer::Vector3D Vector3D;
typedef raytracer::ray ray;

raytracer::Vector3D normalize(raytracer::Vector3D v) {
  double length = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
  return (raytracer::Vector3D){v.x / length, v.y / length, v.z / length};
}

void add_second_spectrum_to_first_one(raytracer::spectrum_of_light *s1,
                                      raytracer::spectrum_of_light s2);
void add_second_spectrum_to_first_one(raytracer::spectrum_of_light *s1,
                                      raytracer::spectrum_of_light s2) {
  for (int i = 0; i < s1->number_of_elements; i++) {
    s1->spectrum[i] += s2.spectrum[i];
  }
}

raytracer::Vector3D give_the_normal_of_a_triangle(
    raytracer::triangle
        input) { // Source:
                 // https://stackoverflow.com/questions/19350792/calculate-normal-of-a-single-raytracer::triangle-in-3d-space

  double Nx = (input.coordinates2.y - input.coordinates1.y) *
                  (input.coordinates3.z - input.coordinates1.z) -
              (input.coordinates2.z - input.coordinates1.z) *
                  (input.coordinates3.y - input.coordinates1.y);
  double Ny = (input.coordinates2.z - input.coordinates1.z) *
                  (input.coordinates3.x - input.coordinates1.x) -
              (input.coordinates2.x - input.coordinates1.x) *
                  (input.coordinates3.z - input.coordinates1.z);
  double Nz = (input.coordinates2.x - input.coordinates1.x) *
                  (input.coordinates3.y - input.coordinates1.y) -
              (input.coordinates2.y - input.coordinates1.y) *
                  (input.coordinates3.x - input.coordinates1.x);

  raytracer::Vector3D to_be_returned;
  to_be_returned.x = Nx;
  to_be_returned.y = Ny;
  to_be_returned.z = Nz;

  return to_be_returned;
}

raytracer::Vector3D copy_given3d(raytracer::Vector3D a) {
  raytracer::Vector3D h;
  h.x = a.x;
  h.y = a.y;
  h.z = a.z;
  return h;
}

raytracer::Vector3D vec3_sub(raytracer::Vector3D a, raytracer::Vector3D b);
raytracer::Vector3D vec3_sub(raytracer::Vector3D a, raytracer::Vector3D b) {
  raytracer::Vector3D result = {a.x - b.x, a.y - b.y, a.z - b.z};
  return result;
}
raytracer::Vector3D cross_p(raytracer::Vector3D a, raytracer::Vector3D b) {
  raytracer::Vector3D result = {a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z,
                                a.x * b.y - a.y * b.x};
  return result;
}

double dot_product(raytracer::Vector3D a, raytracer::Vector3D b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}
bool does_triangle_and_ray_intersect_correctly(
    raytracer::ray R,
    raytracer::triangle
        tri) { // source is //
               // https://stackoverflow.com/questions/42740765/intersection-between-line-and-raytracer::triangle-in-3d
  double t;
  double u;
  double v;
  raytracer::Vector3D N;
  raytracer::Vector3D E1 = vec3_sub(tri.coordinates2, tri.coordinates1);
  raytracer::Vector3D E2 = vec3_sub(tri.coordinates3, tri.coordinates1);

  // printf(" Nx: %f \n", R.ray_current_coordinates.x);
  // printf(" Ny %f \n", R.ray_current_coordinates.y);
  // printf(" Nz %f \n", R.ray_current_coordinates.z);

  N = cross_p(E1, E2);
  double det = -dot_product(R.ray_direction, N);
  if (det < 1e-6)
    return false;
  double invdet = 1.0 / det;
  raytracer::Vector3D AO =
      vec3_sub(R.ray_current_coordinates, tri.coordinates1);
  raytracer::Vector3D DAO = cross_p(AO, R.ray_direction);
  u = dot_product(E2, DAO) * invdet;
  v = -dot_product(E1, DAO) * invdet;
  t = dot_product(AO, N) * invdet;
  // printf("Boolean value: %d\n", (int)(t >= 0 && u >= 0 && v >= 0 && (u + v)
  // <= 1.0));
  return (t >= 0 && u >= 0 && v >= 0 && (u + v) <= 1.0);
}
raytracer::Vector3D intersectLinePlane(raytracer::Vector3D P0,
                                       raytracer::Vector3D v, double A,
                                       double B, double C, double D) {
  raytracer::Vector3D intersection;
  double t =
      (-D - A * P0.x - B * P0.y - C * P0.z) / (A * v.x + B * v.y + C * v.z);
  intersection.x = P0.x + t * v.x;
  intersection.y = P0.y + t * v.y;
  intersection.z = P0.z + t * v.z;
  return intersection;
}
double distanceBetweenPoints(raytracer::Vector3D p1, raytracer::Vector3D p2);
double distanceBetweenPoints(raytracer::Vector3D p1, raytracer::Vector3D p2) {
  double dx = p2.x - p1.x;
  double dy = p2.y - p1.y;
  double dz = p2.z - p1.z;
  return sqrt(dx * dx + dy * dy + dz * dz);
}

double magnitude(raytracer::Vector3D v);
double magnitude(raytracer::Vector3D v) {
  return sqrt((float)(v.x * v.x + v.y * v.y + v.z * v.z));
}

double compute_distance_with_a_triangle_and_ray(raytracer::ray R,
                                                raytracer::triangle tri) {
  raytracer::Vector3D normal = give_the_normal_of_a_triangle(tri);
  double D = -(normal.x * tri.coordinates1.x + normal.y * tri.coordinates1.y +
               normal.z * tri.coordinates1.z);
  raytracer::Vector3D intersection =
      intersectLinePlane(R.ray_current_coordinates, R.ray_direction, normal.x,
                         normal.y, normal.z, D);
  double distance =
      distanceBetweenPoints(intersection, R.ray_current_coordinates);
  double mag = magnitude(R.ray_direction);
  return distance * mag;
}
raytracer::Vector3D vectorFromPoints(raytracer::Vector3D A,
                                     raytracer::Vector3D B) {
  raytracer::Vector3D result;
  result.x = B.x - A.x;
  result.y = B.y - A.y;
  result.z = B.z - A.z;
  return result;
}

// Function to calculate the minimum distance between a point and a line in 3D
double minimumDistance(raytracer::Vector3D A, raytracer::Vector3D d,
                       raytracer::Vector3D P) {

  raytracer::Vector3D lin = vectorFromPoints(A, P);

  raytracer::Vector3D lin_cross_d = cross_p(lin, d);

  double magnitude_lin_cross_d = magnitude(lin_cross_d);
  double magnitude_d = magnitude(d);

  double distance = magnitude_lin_cross_d / magnitude_d;

  double dot = dot_product(lin, d);
  if (dot < 0) {
    distance = -1.0;
  }

  return distance;
}

double
compute_distance_with_a_ray_and_a_point(raytracer::ray current_ray,
                                        raytracer::Vector3D light_position) {

  return minimumDistance(current_ray.ray_current_coordinates,
                         current_ray.ray_direction, light_position);
}
raytracer::Vector3D
find_intersection_of_a_triangle_with_a_ray(raytracer::ray R,
                                           raytracer::triangle tri);
raytracer::Vector3D find_intersection_of_a_triangle_with_a_ray(
    raytracer::ray R,
    raytracer::triangle tri) { // written with the help of ChatGPT
  double x0 = R.ray_current_coordinates.x, y0 = R.ray_current_coordinates.y,
         z0 = R.ray_current_coordinates.z;
  double dx = R.ray_direction.x, dy = R.ray_direction.y, dz = R.ray_direction.z;
  raytracer::Vector3D normal = give_the_normal_of_a_triangle(tri);
  double D1 = -(normal.x * tri.coordinates1.x + normal.y * tri.coordinates1.y +
                normal.z * tri.coordinates1.z);
  double A = normal.x, B = normal.y, C = normal.z, D = D1;

  double denominator = A * dx + B * dy + C * dz;

  double numerator = -(A * x0 + B * y0 + C * z0 + D);
  double t = numerator / denominator;

  raytracer::Vector3D intersection;

  intersection.x = x0 + t * dx;
  intersection.y = y0 + t * dy;
  intersection.z = z0 + t * dz;

  return intersection;
}
Vector3D give_specular_direction(ray current_ray, raytracer::triangle tri);
Vector3D give_specular_direction(ray current_ray, raytracer::triangle tri) {
  Vector3D planeNormal = give_the_normal_of_a_triangle(tri);
  Vector3D incidentRay = current_ray.ray_direction;

  float cosTheta_i = dot_product(incidentRay, planeNormal) /
                     (magnitude(incidentRay) * magnitude(planeNormal));

  // Calculate the reflected ray direction
  Vector3D reflectedRay = {incidentRay.x - 2 * cosTheta_i * planeNormal.x,
                           incidentRay.y - 2 * cosTheta_i * planeNormal.y,
                           incidentRay.z - 2 * cosTheta_i * planeNormal.z};

  // Normalize the reflected ray direction
  float mag = magnitude(reflectedRay);
  reflectedRay.x /= mag;
  reflectedRay.y /= mag;
  reflectedRay.z /= mag;

  return reflectedRay;
}

Vector3D random_direction(Vector3D normal1, uint *seed) {
  // Ensure the normal vector is normalized
  Vector3D normal = normalize(normal1);

  // Generate a random point on the unit hemisphere using spherical coordinates
  double phi = 2.0 * M_PI * random_double_between_0_1_giver(seed);
  double cosTheta = random_double_between_0_1_giver(seed);
  double sinTheta = sqrt(1.0 - cosTheta * cosTheta);

  Vector3D randomDirection = {sinTheta * cos(phi), sinTheta * sin(phi),
                              cosTheta};

  // Create a coordinate system (u, v, w) with w aligned with the normal
  Vector3D w = normal;
  Vector3D u = fabs(w.x) > 0.0001 ? (Vector3D){0, 1, 0} : (Vector3D){1, 0, 0};
  u = normalize(cross_p(u, w));
  Vector3D v = cross_p(w, u);

  // Transform the random direction to align with the normal
  Vector3D result = {randomDirection.x * u.x + randomDirection.y * v.x +
                         randomDirection.z * w.x,
                     randomDirection.x * u.y + randomDirection.y * v.y +
                         randomDirection.z * w.y,
                     randomDirection.x * u.z + randomDirection.y * v.z +
                         randomDirection.z * w.z};

  return result;
}

Vector3D give_diffusive_direction(ray current_ray, raytracer::triangle tri,
                                  uint *seed);
Vector3D give_diffusive_direction(ray current_ray, raytracer::triangle tri,
                                  uint *seed) {
  Vector3D planeNormal = give_the_normal_of_a_triangle(tri);

  return random_direction(planeNormal, seed);
}

raytracer::Vector2D randomPointInRange(raytracer::Vector2D vector, double r,
                                       uint *seed);
raytracer::Vector2D randomPointInRange(raytracer::Vector2D vector, double r,
                                       uint *seed) {

  double angle = 2 * M_PI * random_double_between_0_1_giver(seed);

  double distance = r + (random_double_between_0_1_giver(seed) * r);

  raytracer::Vector2D point;
  point.x = vector.x + distance * cos((float)angle);
  point.y = vector.y + distance * sin((float)angle);

  return point;
}
void array_element_remover(raytracer::Vector2D *array, int index,
                           int array_total_number_of_elemets);
void array_element_remover(raytracer::Vector2D *array, int index,
                           int array_total_number_of_elemets) {
  for (int a = 0; a < array_total_number_of_elemets - index - 1; a++) {
    array[index + a] = array[index + a + 1];
  }
}
void populate_the_random_k_points(raytracer::Vector2D given, double min,
                                  raytracer::Vector2D *to_be_populated,
                                  uint *seed);
void populate_the_random_k_points(raytracer::Vector2D given, double min,
                                  raytracer::Vector2D *to_be_populated,
                                  uint *seed) {
  for (int a = 0; a < 30; a++) {
    to_be_populated[a] = randomPointInRange(given, min, seed);
  }
}
double distance_between(raytracer::Vector2D v1, raytracer::Vector2D v2);
double distance_between(raytracer::Vector2D v1, raytracer::Vector2D v2) {
  return (double)sqrt(
      (float)((v1.x - v2.x) * (v1.x - v2.x) + (v1.y - v2.y) * (v1.y - v2.y)));
}

int randomIntInRange(int n, int m, uint *seed);
int randomIntInRange(int n, int m, uint *seed) {
  return n + (int)(random_double_between_0_1_giver(seed) * (double)(m - n - 1));
}

int fitness_checker(raytracer::Vector2D current_point,
                    raytracer::Vector2D *location_array, int rows, int cols,
                    int background_array[50][50], double length, double width,
                    int m, int n, double r, double tek_kare_uzunluk);
int fitness_checker(raytracer::Vector2D current_point,
                    raytracer::Vector2D *location_array, int rows, int cols,
                    int background_array[50][50], double length, double width,
                    int m, int n, double r, double tek_kare_uzunluk) {
  if (current_point.x > 0 && current_point.x < length && current_point.y > 0 &&
      current_point.y < width) {
    int x0 = my_floor(current_point.x / tek_kare_uzunluk);

    int y0 = my_floor(current_point.y / tek_kare_uzunluk);

    int i0 = fmax(y0 - 1, 0);
    int i1 = fmin(y0 + 1, m - 1);

    int j0 = fmax(x0 - 1, 0);
    int j1 = fmin(x0 + 1, n - 1);

    for (int i = i0; i <= i1; i++) {
      for (int j = j0; j <= j1; j++) {
        if (background_array[i][j] > -1 || (i == y0 && j == x0)) {

          int our_index = background_array[i][j];

          double dista = (sqrt(
              pow((float)(location_array[our_index].x - current_point.x), 2) +
              pow((float)(location_array[our_index].y - current_point.y), 2)));

          if (x0 == 6 && y0 == 1) {
          }

          if (dista < r)
            return 0;
        }
      }
    }

  } else {
    return 0;
  }

  return 1;
}

raytracer::Vector2D_array random_points_giver(double width, double length,
                                              double minimum_distance,
                                              uint *seed);
raytracer::Vector2D_array random_points_giver(double width, double length,
                                              double minimum_distance,
                                              uint *seed) {

  double tek_kare_uzunluk = minimum_distance / sqrt((float)2);
  int n = my_ceil(length / tek_kare_uzunluk);
  int m = my_ceil(width / tek_kare_uzunluk);

  // int background_array[m][n];
  int background_array[50][50];

  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      background_array[i][j] = -1;
    }
  }
  raytracer::Vector2D initial_random_vector;
  initial_random_vector.x = random_double_between_0_1_giver(seed) * length;
  initial_random_vector.y = random_double_between_0_1_giver(seed) * width;

  int x0 = my_floor(initial_random_vector.x / tek_kare_uzunluk);

  int y0 = my_floor(initial_random_vector.y / tek_kare_uzunluk);

  background_array[x0][y0] = 0;

  int active_array_current_index = 0;
  int location_array_index = 0;

  // raytracer::Vector2D active_array[n*m];
  // raytracer::Vector2D location_array[n*m];

  raytracer::Vector2D active_array[50 * 50];
  raytracer::Vector2D location_array[50 * 50];

  active_array[active_array_current_index] = initial_random_vector;
  location_array[location_array_index] = initial_random_vector;
  active_array_current_index++;
  location_array_index += 1;

  while (active_array_current_index > 0) {

    raytracer::Vector2D random_k_points[30];
    int j = randomIntInRange(0, active_array_current_index, seed);

    populate_the_random_k_points(active_array[j], minimum_distance,
                                 random_k_points, seed);
    for (int i = 0; i < 30; i++) {

      raytracer::Vector2D current_point = random_k_points[i];

      int j = fitness_checker(current_point, location_array, m, n,
                              background_array, length, width, m, n,
                              minimum_distance, tek_kare_uzunluk);

      if (j == 1) {
        active_array[active_array_current_index] = current_point;
        active_array_current_index += 1;
        int x1 = my_floor(current_point.x / tek_kare_uzunluk);

        int y1 = my_floor(current_point.y / tek_kare_uzunluk);

        background_array[y1][x1] = location_array_index;

        location_array[location_array_index] = current_point;

        location_array_index += 1;

        break;
      }
      if (i == 29) {
        array_element_remover(active_array, j, active_array_current_index);
        active_array_current_index -= 1;
      }
    }
  }
  raytracer::Vector2D_array to_be_returned;
  to_be_returned.array = location_array;
  to_be_returned.number_of_elements = location_array_index;
  return to_be_returned;
}
raytracer::Vector3D_array
place_it_to_new_location(raytracer::Vector2D_array *input,
                         double new_x_coorinate, double new_y_coordiante);
raytracer::Vector3D_array
place_it_to_new_location(raytracer::Vector2D_array *input,
                         double new_x_coorinate, double new_y_coordiante) {
  raytracer::Vector3D_array to_be_returned;
  memset(to_be_returned.array, 0, sizeof(Vector3D));
  to_be_returned.number_of_elements = input->number_of_elements;
  for (int i = 0; i < input->number_of_elements; i++) {
    Vector3D point;
    point.x = input->array[i].x + new_x_coorinate;
    point.y = input->array[i].y + new_y_coordiante;
    point.z = (double)1;
    to_be_returned.array[i] = point;
  }
  return to_be_returned;
}

// below are the functions to convert a spectrum a an raytracer::RGB value

typedef struct {
  double wavelength;
  double x_bar;
  double y_bar;
  double z_bar;
} CIE1931;
// Placeholder function for CIE 1931 color matching functions
void wavelength_to_xyz(double wavelength, double *x_bar, double *y_bar,
                       double *z_bar);
void wavelength_to_xyz(
    double wavelength, double *x_bar, double *y_bar,
    double *z_bar) { // this function and data are obtained from chatgpt
  // Replace this function with actual CIE 1931 color matching functions
  // Example: return x_bar, y_bar, z_bar for the given wavelength
  // This example uses placeholder values and should be replaced with real data.
  CIE1931 cie_data[] = {
      {380, 0.001368, 0.000039, 0.006450}, {385, 0.002236, 0.000064, 0.010550},
      {390, 0.004243, 0.000120, 0.020050}, {395, 0.007650, 0.000217, 0.036210},
      {400, 0.014310, 0.000396, 0.067850}, {405, 0.023190, 0.000640, 0.110200},
      {410, 0.043510, 0.001210, 0.207400}, {415, 0.077630, 0.002180, 0.371300},
      {420, 0.134380, 0.004000, 0.645600}, {425, 0.214770, 0.007300, 1.039050},
      {430, 0.283900, 0.011600, 1.385600}, {435, 0.328500, 0.016840, 1.622960},
      {440, 0.348280, 0.023000, 1.747060}, {445, 0.348060, 0.029800, 1.782600},
      {450, 0.336200, 0.038000, 1.772110}, {455, 0.318700, 0.048000, 1.744100},
      {460, 0.290800, 0.060000, 1.669200}, {465, 0.251100, 0.073900, 1.528100},
      {470, 0.195360, 0.090980, 1.287640}, {475, 0.142100, 0.112600, 1.041900},
      {480, 0.095640, 0.139020, 0.812950}, {485, 0.057950, 0.169300, 0.616200},
      {490, 0.032010, 0.208020, 0.465180}, {495, 0.014700, 0.258600, 0.353300},
      {500, 0.004900, 0.323000, 0.272000}, {505, 0.002400, 0.407300, 0.212300},
      {510, 0.009300, 0.503000, 0.158200}, {515, 0.029100, 0.608200, 0.111700},
      {520, 0.063270, 0.710000, 0.078250}, {525, 0.109600, 0.793200, 0.057250},
      {530, 0.165500, 0.862000, 0.042160}, {535, 0.225750, 0.914850, 0.029840},
      {540, 0.290400, 0.954000, 0.020300}, {545, 0.359700, 0.980300, 0.013400},
      {550, 0.433450, 0.994950, 0.008750}, {555, 0.512050, 1.000000, 0.005750},
      {560, 0.594500, 0.995000, 0.003900}, {565, 0.678400, 0.978600, 0.002750},
      {570, 0.762100, 0.952000, 0.002100}, {575, 0.842500, 0.915400, 0.001800},
      {580, 0.916300, 0.870000, 0.001650}, {585, 0.978600, 0.816300, 0.001400},
      {590, 1.026300, 0.757000, 0.001100}, {595, 1.056700, 0.694900, 0.001000},
      {600, 1.062200, 0.631000, 0.000800}, {605, 1.045600, 0.566800, 0.000600},
      {610, 1.002600, 0.503000, 0.000340}, {615, 0.938400, 0.441200, 0.000240},
      {620, 0.854450, 0.381000, 0.000190}, {625, 0.751400, 0.321000, 0.000100},
      {630, 0.642400, 0.265000, 0.000050}, {635, 0.541900, 0.217000, 0.000030},
      {640, 0.447900, 0.175000, 0.000020}, {645, 0.360800, 0.138200, 0.000010},
      {650, 0.283500, 0.107000, 0.000000}, {655, 0.218700, 0.081600, 0.000000},
      {660, 0.164900, 0.061000, 0.000000}, {665, 0.121200, 0.044580, 0.000000},
      {670, 0.087400, 0.032000, 0.000000}, {675, 0.063600, 0.023200, 0.000000},
      {680, 0.046770, 0.017000, 0.000000}, {685, 0.032900, 0.011920, 0.000000},
      {690, 0.022700, 0.008210, 0.000000}, {695, 0.015840, 0.005723, 0.000000},
      {700, 0.011359, 0.004102, 0.000000},
  };
  // Find the closest wavelength in the CIE data
  int i = 0;
  while (cie_data[i].wavelength < wavelength) {
    i++;
    if (i >= sizeof(cie_data) / sizeof(cie_data[0])) {
      i = sizeof(cie_data) / sizeof(cie_data[0]) - 1;
      break;
    }
  }
  // the following 15 lines are taken from ChatGPT
  // Interpolate to get the corresponding x_bar, y_bar, z_bar
  if (i == 0) {
    *x_bar = cie_data[0].x_bar;
    *y_bar = cie_data[0].y_bar;
    *z_bar = cie_data[0].z_bar;
  } else if (i >= sizeof(cie_data) / sizeof(cie_data[0])) {
    *x_bar = cie_data[sizeof(cie_data) / sizeof(cie_data[0]) - 1].x_bar;
    *y_bar = cie_data[sizeof(cie_data) / sizeof(cie_data[0]) - 1].y_bar;
    *z_bar = cie_data[sizeof(cie_data) / sizeof(cie_data[0]) - 1].z_bar;
  } else {
    double ratio = (wavelength - cie_data[i - 1].wavelength) /
                   (cie_data[i].wavelength - cie_data[i - 1].wavelength);
    *x_bar = cie_data[i - 1].x_bar +
             (cie_data[i].x_bar - cie_data[i - 1].x_bar) * ratio;
    *y_bar = cie_data[i - 1].y_bar +
             (cie_data[i].y_bar - cie_data[i - 1].y_bar) * ratio;
    *z_bar = cie_data[i - 1].z_bar +
             (cie_data[i].z_bar - cie_data[i - 1].z_bar) * ratio;
  }
}

// the following 4 functions ro convert a spectrum to an raytracer::RGB value
// are taken from ChatGPT
void spectral_to_xyz(double *spd, double *wavelengths, int length, double *X,
                     double *Y, double *Z);
void spectral_to_xyz(double *spd, double *wavelengths, int length, double *X,
                     double *Y, double *Z) {
  // Normalize SPD values if not already normalized

  double max_spd = 0.0;
  for (int i = 0; i < length; ++i) {
    if (spd[i] > max_spd) {

      max_spd = spd[i];
    }
  }
  if (max_spd > 1.0) {
    for (int i = 0; i < length; ++i) {
      spd[i] /= max_spd;
    }
  }

  // Initialize XYZ values
  *X = 0;
  *Y = 0;
  *Z = 0;
  for (int i = 0; i < length; ++i) {
    if (spd[i] != spd[i]) {

    } else {

      double lambda = wavelengths[i];
      double x_bar, y_bar, z_bar;
      wavelength_to_xyz(lambda, &x_bar, &y_bar, &z_bar);
      *X += spd[i] * x_bar;
      *Y += spd[i] * y_bar;
      *Z += spd[i] * z_bar;
    }
  }
}
void xyz_to_RGB(double X, double Y, double Z, double *R, double *G, double *B);
void xyz_to_RGB(double X, double Y, double Z, double *R, double *G, double *B) {
  // Define the transformation matrix from XYZ to linear sraytracer::RGB

  double M[3][3] = {{3.2406, -1.5372, -0.4986},
                    {-0.9689, 1.8758, 0.0415},
                    {0.0557, -0.2040, 1.0570}};
  *R = M[0][0] * X + M[0][1] * Y + M[0][2] * Z;
  *G = M[1][0] * X + M[1][1] * Y + M[1][2] * Z;
  *B = M[2][0] * X + M[2][1] * Y + M[2][2] * Z;
}
double gamma_correct(double value);
double gamma_correct(double value) {

  if (value <= 0.0031308) {
    return 12.92 * value;
  } else {
    return 1.055 * pow((float)value, 1 / 2.4) - 0.055;
  }
}
void spectral_to_RGB(double *spd, double *wavelengths, int length, int *R,
                     int *G, int *B);
void spectral_to_RGB(double *spd, double *wavelengths, int length, int *R,
                     int *G, int *B) {
  // Convert the spectral distribution to XYZ
  double X, Y, Z;
  spectral_to_xyz(spd, wavelengths, length, &X, &Y, &Z);

  // Convert XYZ to linear raytracer::RGB
  double RGB_linear[3];
  xyz_to_RGB(X, Y, Z, &RGB_linear[0], &RGB_linear[1], &RGB_linear[2]);

  // Apply gamma correction
  double RGB_corrected[3];
  for (int i = 0; i < 3; ++i) {
    RGB_corrected[i] = gamma_correct(RGB_linear[i]);
  }

  // Clamp and scale to [0, 255]

  *R = (int)(fmax((float)0.0, fmin((float)1.0, RGB_corrected[0])) * 255);
  *G = (int)(fmax((float)0.0, fmin((float)1.0, RGB_corrected[1])) * 255);
  *B = (int)(fmax((float)0.0, fmin((float)1.0, RGB_corrected[2])) * 255);
}

// ray tracing computations are below
raytracer::spectrum_of_light
compute_the_ray(ray current_ray, raytracer::surface_array *all_surfaces,
                raytracer::triangle_array *all_faces, Vector3D light_position,
                double light_radius, uint *seed, int max_recursion);
raytracer::spectrum_of_light
compute_the_ray(ray current_ray, raytracer::surface_array *all_surfaces,
                raytracer::triangle_array *all_faces, Vector3D light_position,
                double light_radius, uint *seed, int max_recursion) {

  if (current_ray.recursion_index + 1 > max_recursion) {

    return current_ray.ray_spectrum;
  }

  int triangle_index = -1;
  double minimum_distance_of_ray_to_a_traingle = 999999999;

  for (int a = 0; a < all_faces->total_number_of_triangles; a++) {
    if (does_triangle_and_ray_intersect_correctly(
            current_ray, all_faces->all_triangles[a])) {
      double distance = compute_distance_with_a_triangle_and_ray(
          current_ray, all_faces->all_triangles[a]);
      // printf(" distance %f  \n", distance);
      if (a == 0 || (distance < minimum_distance_of_ray_to_a_traingle)) {
        triangle_index = a;

        minimum_distance_of_ray_to_a_traingle = distance;
      }
    }
  }

  double distance_to_light =
      compute_distance_with_a_ray_and_a_point(current_ray, light_position);

  // it directly intersects with the light

  if (distance_to_light <= light_radius && distance_to_light >= 0) {

    return current_ray.ray_spectrum;
  }

  if (triangle_index == -1) {
    raytracer::spectrum_of_light zero_one;
    zero_one.number_of_elements = (700 - 380) / 5 + 1;
    for (int y = 0; y < zero_one.number_of_elements; y++) {
      zero_one.spectrum[y] = 0;
    }
    return zero_one;
  }

  ray new_ray_specular;
  raytracer::spectrum_of_light spectrum_of_current_ray;
  spectrum_of_current_ray.number_of_elements = (700 - 380) / 5 + 1;

  int surface_index = all_faces->all_triangles[triangle_index].surface_number;

  for (int o = 0; o < spectrum_of_current_ray.number_of_elements; o++) {
    double a1 = current_ray.ray_spectrum.spectrum[o];
    raytracer::surface a2 = all_surfaces->surface_array[surface_index];
    double a3 = a2.surface_spectrum.spectrum[o];
    double a4 =
        (1 -
         (all_surfaces->surface_array[surface_index]).diffusion_coefficient);

    spectrum_of_current_ray.spectrum[o] = a1 * a3 * a4;
  }
  new_ray_specular.ray_spectrum = spectrum_of_current_ray;

  ray new_ray_diffusion;
  raytracer::spectrum_of_light spectrum_of_current_ray2;
  spectrum_of_current_ray2.number_of_elements = (700 - 380) / 5 + 1;

  for (int o = 0; o < spectrum_of_current_ray2.number_of_elements; o++) {
    double a1 = current_ray.ray_spectrum.spectrum[o];
    raytracer::surface a2 = all_surfaces->surface_array[surface_index];
    double a3 = a2.surface_spectrum.spectrum[o];
    double a4 =
        ((all_surfaces->surface_array[surface_index]).diffusion_coefficient);
    spectrum_of_current_ray2.spectrum[o] = a1 * a3 * a4;
  }

  new_ray_diffusion.ray_spectrum = spectrum_of_current_ray2;
  Vector3D new_location = find_intersection_of_a_triangle_with_a_ray(
      current_ray, all_faces->all_triangles[triangle_index]);
  Vector3D new_specular_direction = give_specular_direction(
      current_ray, all_faces->all_triangles[triangle_index]);
  Vector3D new_diffusive_direction = give_diffusive_direction(
      current_ray, all_faces->all_triangles[triangle_index], seed);

  new_ray_specular.ray_current_coordinates = new_location;
  new_ray_diffusion.ray_current_coordinates = new_location;

  new_ray_specular.ray_direction = new_specular_direction;
  new_ray_diffusion.ray_direction = new_diffusive_direction;

  new_ray_specular.recursion_index = current_ray.recursion_index + 1;
  new_ray_diffusion.recursion_index = current_ray.recursion_index + 1;

  raytracer::spectrum_of_light specular_spectrum =
      compute_the_ray(new_ray_specular, all_surfaces, all_faces, light_position,
                      light_radius, seed, max_recursion);
  raytracer::spectrum_of_light diffusive_spectrum =
      compute_the_ray(new_ray_diffusion, all_surfaces, all_faces,
                      light_position, light_radius, seed, max_recursion);

  add_second_spectrum_to_first_one(&specular_spectrum, diffusive_spectrum);

  return specular_spectrum;
}

raytracer::RGB trace(int which_pixel_x_coord, int which_pixel_y_coord,
                     double camara_plane_x_width, double camera_plane_y_width,
                     int number_of_x_pixels, int number_of_y_pixels,
                     int random_number_generator_seed, double minimum_distance,
                     raytracer::surface_array all_surfaces,
                     raytracer::triangle_array all_faces,
                     Vector3D light_position, double light_radius,
                     int max_recursion) {
  uint our_random_number = (uint)random_number_generator_seed;

  raytracer::spectrum_of_light resultant_spectrum;
  resultant_spectrum.number_of_elements = (700 - 380) / 5 + 1;
  for (int t = 0; t < resultant_spectrum.number_of_elements; t++) {
    resultant_spectrum.spectrum[t] = 0.0;
  }

  raytracer::Vector2D_array to_be_returned =
      random_points_giver(camera_plane_y_width / ((double)(number_of_y_pixels)),
                          camara_plane_x_width / ((double)(number_of_x_pixels)),
                          minimum_distance, &our_random_number);

  // printf("The integer final final is: %d\n",
  // to_be_returned.number_of_elements);

  raytracer::Vector3D_array final_plane =
      place_it_to_new_location(&to_be_returned,
                               which_pixel_x_coord * camara_plane_x_width /
                                       ((double)(number_of_x_pixels)) -
                                   (camara_plane_x_width / 2),
                               which_pixel_y_coord * camera_plane_y_width /
                                       ((double)(number_of_y_pixels)) -
                                   (camera_plane_y_width / 2));

  for (int i = 0; i < final_plane.number_of_elements; i++) {

    ray current_ray;
    current_ray.recursion_index = 0;
    raytracer::spectrum_of_light spectrum_of_current_ray;
    spectrum_of_current_ray.number_of_elements = (700 - 380) / 5 + 1;
    // initialize the white light

    for (int o = 0; o < spectrum_of_current_ray.number_of_elements; o++) {
      spectrum_of_current_ray.spectrum[o] = (double)1;
    }
    current_ray.ray_spectrum = spectrum_of_current_ray;
    current_ray.ray_direction = copy_given3d(
        final_plane.array[i]); // The COP is assumed to be at the origin
    current_ray.ray_current_coordinates = copy_given3d(final_plane.array[i]);

    raytracer::spectrum_of_light the_new_spectrum =
        compute_the_ray(current_ray, &all_surfaces, &all_faces, light_position,
                        light_radius, &our_random_number, max_recursion);

    add_second_spectrum_to_first_one(&resultant_spectrum, the_new_spectrum);
  }

  for (int j = 0; j < resultant_spectrum.number_of_elements; j++) {
    resultant_spectrum.spectrum[j] /= (double)final_plane.number_of_elements;
  }

  // code to convert the final resultant light spectrum into raytracer::RGB
  // values
  int length = (int)((700 - 380) / 5 + 1);
  double wavelengths[length];
  double spd[length];

  for (int i = 0; i < length; ++i) {
    wavelengths[i] =
        380 + 5 * i; // Example wavelengths from 380nm to 780nm in 5nm steps
    spd[i] = resultant_spectrum.spectrum[i];
    if (spd[i] != 0.0) {
      // printf("spectrum %f \n",spd[i]);
    }
  }

  int R, G, B;

  spectral_to_RGB(spd, wavelengths, length, &R, &G, &B);
  // printf("raytracer::RGB: (%d, %d, %d)\n", R, G, B);

  raytracer::RGB colour;
  colour.R = (uchar)R;
  colour.G = (uchar)G;
  colour.B = (uchar)B;

  return colour;
}

#define VEC3TOFLOAT4ONEIDX(vec3, float4, i) float4.i = vec3.i
#define VEC3TOFLOAT4(vec3, float4)                                             \
  VEC3TOFLOAT4ONEIDX(vec3, float4, x);                                         \
  VEC3TOFLOAT4ONEIDX(vec3, float4, y);                                         \
  VEC3TOFLOAT4ONEIDX(vec3, float4, z);

#define CHECK_ERROR(x) CL_RUN3(;)

#define RANDOM_POINTS_GIVER_ARRAY_ONE_DIM (2)
#define RANDOM_POINTS_GIVER_ARRAY_LENGTH                                       \
  (RANDOM_POINTS_GIVER_ARRAY_ONE_DIM * RANDOM_POINTS_GIVER_ARRAY_ONE_DIM)

int main() {

  raytracer::surface yellow_surface;
  raytracer::spectrum_of_light yellow_spectrum;
  yellow_spectrum.number_of_elements = (int)((700 - 380) / 5 + 1);
  for (int a = 380; a < 700 + 1; a += 5) {
    if (a >= 560 && a <= 590) {
      yellow_spectrum.spectrum[(int)((a - 380) / 5)] = 0.9;

    } else {
      yellow_spectrum.spectrum[(int)((a - 380) / 5)] = 0.1;
    }
  }
  yellow_surface.diffusion_coefficient = 0.6;

  raytracer::surface_array all_surfaces;
  yellow_surface.surface_spectrum = yellow_spectrum;
  all_surfaces.surface_array[0] = yellow_surface;

  raytracer::triangle yellow_triangle;
  yellow_triangle.surface_number = 0;
  Vector3D v1;
  v1.x = -1.0;
  v1.y = 0.0;
  v1.z = 3.0;
  Vector3D v2;
  v2.x = 0.0;
  v2.y = 1.0;
  v2.z = 3.0;
  Vector3D v3;
  v3.x = 1.0;
  v3.y = 0.0;
  v3.z = 3.0;

  yellow_triangle.coordinates1 = v1;
  yellow_triangle.coordinates2 = v2;
  yellow_triangle.coordinates3 = v3;

  raytracer::triangle_array all_tris;
  all_tris.total_number_of_triangles = 1;
  all_tris.all_triangles[0] = yellow_triangle;

  Vector3D light_position;
  light_position.x = 0.0;
  light_position.y = 0.0;
  light_position.z = 0.0;

  /**
  __kernel void trace(
    double camara_plane_x_width, double camera_plane_y_width,
    int number_of_x_pixels, int number_of_y_pixels,
    int random_number_generator_seed, double minimum_distance,
    __global double *all_surfaces_spectrums, __global double
  *all_surfaces_coeffs, int surface_count,
    __global float4 *all_triangles_coords, __global int *all_triangles_surfaces,
  int triangle_count, float4 light_position, double light_radius, int
  max_recursion, __global float4 *out_data)
  */

  int num_surfaces = 4; // I guess hard coded 50?
  int num_triangles = all_tris.total_number_of_triangles;
  printf("NUM TRIANGLES: %d\n", num_triangles);
  float *all_surfaces_spectrums;
  float *all_surfaces_coeffs;
  float4 *all_triangles_coords;
  int *all_triangles_surfaces;

  all_surfaces_spectrums =
      (float *)calloc(num_surfaces, sizeof(float) * SPECTRUM_SIZE);
  all_surfaces_coeffs = (float *)calloc(num_surfaces, sizeof(float));
  all_triangles_coords = (float4 *)calloc(num_triangles, sizeof(float4) * 3);
  all_triangles_surfaces = (int *)calloc(num_triangles, sizeof(int));

  float4 *outc = (float4 *)calloc(OUT_BUFFER_LEN, sizeof(float4));

  // put all the data in

  for (int i = 0; i < all_tris.total_number_of_triangles; i++) {
    auto tri = all_tris.all_triangles + i;
    VEC3TOFLOAT4(tri->coordinates1, all_triangles_coords[(i * 3) + 0]);
    VEC3TOFLOAT4(tri->coordinates2, all_triangles_coords[(i * 3) + 1]);
    VEC3TOFLOAT4(tri->coordinates3, all_triangles_coords[(i * 3) + 2]);
    all_triangles_surfaces[i] = tri->surface_number;
  }

  for (int i = 0; i < num_surfaces; i++) {
    auto surface = all_surfaces.surface_array + i;
    for (int j = 0; j < SPECTRUM_SIZE; j++) {
      all_surfaces_spectrums[(i * SPECTRUM_SIZE) + j] =
          surface->surface_spectrum.spectrum[j];
    }
    all_surfaces_coeffs[i] = surface->diffusion_coefficient;
  }

  double cam_w = 1.0;
  double cam_h = 1.0;

  int num_pixels[] = {NUM_PIXELS_X, NUM_PIXELS_Y};

  int seed = 42;

  double min_distance = 0.0005;

  double light_radius = 2.0;

  int max_recursion = 6;

  float4 light_pos4;

  VEC3TOFLOAT4(light_position, light_pos4);

  // CL_RUN3(cl_mem d_all_surfaces_spectrums =
  //             clCreateBuffer(context, CL_MEM_READ_ONLY |
  //             CL_MEM_COPY_HOST_PTR,
  //                            num_surfaces * SPECTRUM_SIZE * sizeof(float),
  //                            all_surfaces_spectrums, &err));
  // CL_RUN3(cl_mem d_all_surfaces_coeffs = clCreateBuffer(
  //             context, CL_MEM_READ_ONLY | CL_MEM_COPY_HOST_PTR,
  //             num_surfaces * sizeof(float), all_surfaces_coeffs, &err);)
  // CL_RUN3(cl_mem d_all_triangles_coords = clCreateBuffer(
  //             context, CL_MEM_READ_ONLY | CL_MEM_COPY_HOST_PTR,
  //             num_triangles * 3 * sizeof(float4), all_triangles_coords,
  //             &err);)
  // CL_RUN3(cl_mem d_all_triangles_surfaces = clCreateBuffer(
  //             context, CL_MEM_READ_ONLY | CL_MEM_COPY_HOST_PTR,
  //             num_triangles * sizeof(cl_int), all_triangles_surfaces, &err));
  // CL_RUN3(cl_mem d_out_data =
  //             clCreateBuffer(context, CL_MEM_READ_WRITE |
  //             CL_MEM_COPY_HOST_PTR,
  //                            OUT_BUFFER_LEN * sizeof(float4), outc, &err););

  int randsize = RANDOM_POINTS_GIVER_ARRAY_LENGTH * OUT_BUFFER_LEN;
  // CL_RUN3(cl_mem d_g_to_be_returned =
  //             clCreateBuffer(context, CL_MEM_READ_WRITE |
  //             CL_MEM_HOST_NO_ACCESS,
  //                            randsize * sizeof(float2), NULL, &err););
  // CL_RUN3(cl_mem d_g_final_plane =
  //             clCreateBuffer(context, CL_MEM_READ_WRITE |
  //             CL_MEM_HOST_NO_ACCESS,
  //                            randsize * sizeof(float4), NULL, &err););
  // CL_RUN3(cl_mem d_random_buffer =
  //             clCreateBuffer(context, CL_MEM_READ_WRITE |
  //             CL_MEM_HOST_NO_ACCESS,
  //                            randsize * sizeof(float2) * 2, NULL, &err););
  // CL_RUN3(cl_mem d_bg_buffer =
  //             clCreateBuffer(context, CL_MEM_READ_WRITE |
  //             CL_MEM_HOST_NO_ACCESS,
  //                            randsize * sizeof(cl_int), NULL, &err););
  // // SPECTRUM_NUM_ELEMS
  // CL_RUN3(cl_mem g_all_ray_elems = clCreateBuffer(
  //             context, CL_MEM_READ_WRITE | CL_MEM_HOST_NO_ACCESS,
  //             SPECTRUM_SIZE * OUT_BUFFER_LEN * sizeof(float), NULL, &err););

  // err = clSetKernelArg(kernel, 0, sizeof(float), &cam_w);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 1, sizeof(float), &cam_h);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 2, sizeof(cl_int), &num_pixels[0]);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 3, sizeof(cl_int), &num_pixels[1]);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 4, sizeof(cl_int), &seed);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 5, sizeof(float), &min_distance);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 6, sizeof(cl_mem), &d_all_surfaces_spectrums);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 7, sizeof(cl_mem), &d_all_surfaces_coeffs);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 8, sizeof(cl_int), &num_surfaces);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 9, sizeof(cl_mem), &d_all_triangles_coords);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 10, sizeof(cl_mem),
  // &d_all_triangles_surfaces); CHECK_ERROR(err); err = clSetKernelArg(kernel,
  // 11, sizeof(cl_int), &num_triangles); CHECK_ERROR(err); err =
  // clSetKernelArg(kernel, 12, sizeof(float4), &light_pos4); CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 13, sizeof(float), &light_radius);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 14, sizeof(cl_int), &max_recursion);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 15, sizeof(cl_mem), &d_out_data);
  // CHECK_ERROR(err);

  /**
 __global float2 *g_to_be_returned, __global float4 *g_final_plane, __global
 float2 *random_buffer , __global int *bgbuffer
 */
  // err = clSetKernelArg(kernel, 16, sizeof(cl_mem), &d_g_to_be_returned);
  // CHECK_ERROR(err);

  // err = clSetKernelArg(kernel, 17, sizeof(cl_mem), &d_g_final_plane);
  // CHECK_ERROR(err);

  // err = clSetKernelArg(kernel, 18, sizeof(cl_mem), &d_random_buffer);
  // CHECK_ERROR(err);

  // err = clSetKernelArg(kernel, 19, sizeof(cl_mem), &d_bg_buffer);
  // CHECK_ERROR(err);
  // err = clSetKernelArg(kernel, 20, sizeof(cl_mem), &g_all_ray_elems);
  // CHECK_ERROR(err);

  size_t global_work_size[2] = {NUM_PIXELS_X, NUM_PIXELS_Y};
  size_t local_size[2] = {4, 4};

  // CL_RUN2(clEnqueueNDRangeKernel(queue, kernel, 2, NULL, global_work_size,
  //                                local_size, 0, NULL, NULL));
  // CL_RUN2(clFlush(queue));
  // CL_RUN2(clFinish(queue));

  // CL_RUN2(clEnqueueReadBuffer(queue, d_out_data, CL_TRUE, 0, 1, outc, 0,
  // NULL,
  //                             NULL));

  ray exp;
  Vector3D e = {0.0, 0.01, 1.0};
  exp.ray_current_coordinates = e;
  exp.ray_direction = e;

  save_png("./test.png", (unsigned char *)outc, NUM_PIXELS_X, NUM_PIXELS_Y);

  return 0;
}
